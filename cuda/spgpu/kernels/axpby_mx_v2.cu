#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
  //#include "cuda_util.h"
}


#include "debug.h"

#define BLOCK_SIZE 512


#if 1
__global__ void spgpu_axpby_mx_v2_krn(float *z, int n, float beta, float *y, float alpha, double* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	if (beta == 0.0f) {
	  for ( ; id < n; id +=gridSize)
	    {
	      	// Since z, x and y are accessed with the same offset by the same thread,
	      	// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).
	      
	      	z[id] = alpha * static_cast<float>(x[id]);
	    }
	} else {
	  for ( ; id < n; id +=gridSize)
	    {
	      	// Since z, x and y are accessed with the same offset by the same thread,
	      	// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).
	    	z[id] = alpha * static_cast<float>(x[id]) + beta * y[id];
		}
	}
}

void spgpu_axpby_mx_v2(spgpuHandle_t handle,
	__device float *z,
	int n,
	float beta,
	__device float *y,
	float alpha,
	__device double *x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpu_axpby_mx_v2_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

#else

__global__ void spgpu_axpby_mx_v2_krn(float *z, int n, float beta, float *y, float alpha, double* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < n)
	{
		// Since z, x and y are accessed with the same offset by the same thread,
		// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).

		if (beta == 0.0f)
			z[id] = PREC_FMUL(alpha,x[id]);
		else
	        z[id] = PREC_ADD_MIXED(PREC_FMUL(alpha, x[id]), PREC_MUL_MIXED(beta,y[id]));
	}
}



void spgpu_axpby_mx_v2_(spgpuHandle_t handle,
	__device double *z,
	int n,
	float beta,
	__device double *y,
	float alpha,
	__device float* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	spgpu_axpby_mx_v2_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

void spgpu_axpby_mx_v2(spgpuHandle_t handle,
	__device double *z,
	int n,
	float beta,
	__device double *y,
	float alpha,
	__device float* x)
{
	int maxNForACall = max(handle->maxGridSizeX, BLOCK_SIZE*handle->maxGridSizeX);
	while (n > maxNForACall) //managing large vectors
	{
		spgpu_axpby_mx_v2_(handle, z, maxNForACall, beta, y, alpha, x);
		
		x = x + maxNForACall;
		y = y + maxNForACall;
		z = z + maxNForACall;
		n -= maxNForACall;
	}
	
	spgpu_axpby_mx_v2_(handle, z, n, beta, y, alpha, x);

	cudaCheckError("CUDA error on saxpby");
}
#endif
void spgpuSmaxpby_mx_v2(spgpuHandle_t handle,
		  __device float *z,
		  int n,
		  float beta,
		  __device float *y,
		  float alpha,
		  __device double *x, 
		  int count, int pitch)
{

  for (int i=0; i<count; i++)
    spgpu_axpby_mx_v2(handle, z+pitch*i, n, beta, y+pitch*i, alpha, x+pitch*i);
  
}
