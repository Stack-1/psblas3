#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
  //#include "cuda_util.h"
}


#include "debug.h"

#define BLOCK_SIZE 512


#if 1
__global__ void spgpu_axpby_mx_krn(double *z, int n, float beta, double *y, float alpha, float* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	if (beta == 0.0f) {
	  for ( ; id < n; id +=gridSize)
	    {
	      	// Since z, x and y are accessed with the same offset by the same thread,
	      	// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).
	      
	      	z[id] = static_cast<double>(alpha) * static_cast<double>(x[id]);
	    }
	} else {
	  for ( ; id < n; id +=gridSize)
	    {
	      	// Since z, x and y are accessed with the same offset by the same thread,
	      	// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).
	    	z[id] = static_cast<double>(alpha) * static_cast<double>(x[id]) + static_cast<double>(beta) * y[id];
		}
	}
}

void spgpu_axpby_mx(spgpuHandle_t handle,
	__device double *z,
	int n,
	float beta,
	__device double *y,
	float alpha,
	__device float* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpu_axpby_mx_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

#else

__global__ void spgpu_axpby_mx_krn(double *z, int n, float beta, double *y, float alpha, float* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < n)
	{
		// Since z, x and y are accessed with the same offset by the same thread,
		// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).

		if (beta == 0.0f)
			z[id] = PREC_FMUL(alpha,x[id]);
		else
	        z[id] = PREC_ADD_MIXED(PREC_FMUL(alpha, x[id]), PREC_MUL_MIXED(beta,y[id]));
	}
}



void spgpu_axpby_mx_(spgpuHandle_t handle,
	__device double *z,
	int n,
	float beta,
	__device double *y,
	float alpha,
	__device float* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	spgpu_axpby_mx_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

void spgpu_axpby_mx(spgpuHandle_t handle,
	__device double *z,
	int n,
	float beta,
	__device double *y,
	float alpha,
	__device float* x)
{
	int maxNForACall = max(handle->maxGridSizeX, BLOCK_SIZE*handle->maxGridSizeX);
	while (n > maxNForACall) //managing large vectors
	{
		spgpu_axpby_mx_(handle, z, maxNForACall, beta, y, alpha, x);
		
		x = x + maxNForACall;
		y = y + maxNForACall;
		z = z + maxNForACall;
		n -= maxNForACall;
	}
	
	spgpu_axpby_mx_(handle, z, n, beta, y, alpha, x);

	cudaCheckError("CUDA error on saxpby");
}
#endif
void spgpuSmaxpby_mx(spgpuHandle_t handle,
		  __device double *z,
		  int n,
		  float beta,
		  __device double *y,
		  float alpha,
		  __device float* x, 
		  int count, int pitch)
{

  for (int i=0; i<count; i++)
    spgpu_axpby_mx(handle, z+pitch*i, n, beta, y+pitch*i, alpha, x+pitch*i);
  
}
